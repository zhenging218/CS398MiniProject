#include "hip/hip_runtime.h"
#include "precomp.cuh"
#include "bitboard.h"
#include "gpuminimax.h"

#include <cstdlib>
#include <algorithm>

namespace Checkers
{
	namespace GPUMinimax
	{
		__host__ __device__ Minimax::utility_type BlackMoveMax(BitBoard const &b, int depth, int turns_left, Minimax::utility_type alpha, Minimax::utility_type beta)
		{
			utility_type v = -Infinity;
			utility_type terminal_value = 0;
			// check if need to stop the search
			if (GetBlackUtility(b, terminal_value, depth, turns_left))
				return terminal_value;

			BitBoard frontier[32];
			BitBoard *end = frontier;
			BitBoard::GetPossibleBlackMoves(b, end);
			int size = end - frontier;

			if (size > 0)
			{
				v = max(BlackMoveMin(frontier[0], depth - 1, turns_left - 1, alpha, beta), v);
				if (!(v > beta))
				{
					alpha = std::max(alpha, v);
					if (size > 1)
					{
						GPUBitBoard * GPUFrontier;
						utility_type * GPUv;

						GPUBitBoard *copy = (GPUBitBoard*)malloc(sizeof(GPUBitBoard) * (size - 1));
						for (int i = 0; i < (size - 1); ++i)
						{
							new (copy + i) GPUBitBoard(frontier[i + 1]);
						}
						hipMalloc((void**)&GPUFrontier, sizeof(GPUBitBoard) * (size - 1));
						hipMemcpy(GPUFrontier, copy, sizeof(GPUBitBoard) * (size - 1), hipMemcpyHostToDevice);
						free(copy);

						hipMalloc((void**)&GPUv, sizeof(utility_type));
						hipMemcpy(GPUv, &v, sizeof(utility_type), hipMemcpyHostToDevice);

						// launch kernel
						master_black_max_kernel(GPUv, GPUFrontier, size - 1, alpha, beta, depth - 1, turns_left - 1);
						CHECK_ERRORS();

						hipMemcpy(&v, GPUv, sizeof(int), hipMemcpyDeviceToHost);
						hipFree(GPUFrontier);
						CHECK_ERRORS();
						hipFree(GPUv);
						CHECK_ERRORS();

					}
				}
			}

			return v;
		}

		__host__ __device__ Minimax::utility_type BlackMoveMin(BitBoard const &b, int depth, int turns_left, Minimax::utility_type alpha, Minimax::utility_type beta)
		{
			utility_type v = Infinity;
			utility_type terminal_value = 0;
			// check if need to stop the search
			if (GetBlackUtility(b, terminal_value, depth, turns_left))
				return terminal_value;

			BitBoard frontier[32];
			BitBoard *end = frontier;
			BitBoard::GetPossibleWhiteMoves(b, end);
			int size = end - frontier;

			if (size > 0)
			{
				v = min(BlackMoveMax(frontier[0], depth - 1, turns_left - 1, alpha, beta), v);
				if (!(v < alpha))
				{
					beta = std::min(beta, v);
					if (size > 1)
					{
						GPUBitBoard * GPUFrontier;
						utility_type * GPUv;

						GPUBitBoard *copy = (GPUBitBoard*)malloc(sizeof(GPUBitBoard) * (size - 1));
						for (int i = 0; i < (size - 1); ++i)
						{
							new (copy + i) GPUBitBoard(frontier[i + 1]);
						}
						hipMalloc((void**)&GPUFrontier, sizeof(GPUBitBoard) * (size - 1));
						CHECK_ERRORS();
						hipMemcpy(GPUFrontier, copy, sizeof(GPUBitBoard) * (size - 1), hipMemcpyHostToDevice);
						CHECK_ERRORS();
						free(copy);

						hipMalloc((void**)&GPUv, sizeof(utility_type));
						CHECK_ERRORS();
						hipMemcpy(GPUv, &v, sizeof(utility_type), hipMemcpyHostToDevice);
						CHECK_ERRORS();

						// launch kernel
						master_black_min_kernel(GPUv, GPUFrontier, size - 1, alpha, beta, depth - 1, turns_left - 1);
						CHECK_ERRORS();

						hipMemcpy(&v, GPUv, sizeof(int), hipMemcpyDeviceToHost);
						CHECK_ERRORS();
						hipFree(GPUFrontier);
						CHECK_ERRORS();
						hipFree(GPUv);
						CHECK_ERRORS();

					}
				}
			}

			return v;
		}
	}
}