#include "hip/hip_runtime.h"
#include "precomp.cuh"
#include "bitboard.h"
#include "gpuminimax.h"

namespace Checkers
{
	namespace GPUMinimax
	{
		__global__ void master_white_max_kernel(Minimax::utility_type *v, GPUBitBoard const *src, int num_boards, int alpha, int beta, int depth, int turns)
		{
			int tx = threadIdx.x;
			Minimax::utility_type t_v = *v;
			__shared__ Minimax::utility_type *utility;
			__shared__ Minimax::utility_type t_utility[32];

			if (tx == 0)
			{
				hipMalloc(&utility, sizeof(Minimax::utility_type) * num_boards);
				memset(utility, t_v, sizeof(Minimax::utility_type) * num_boards);
			}

			__syncthreads();

			if (tx < num_boards)
			{
				white_min_kernel << <dim3(1, 1, 1), dim3(32, 1, 1) >> > (utility + tx, src[tx], alpha, beta, depth - 1, turns - 1);
				t_utility[tx] = utility[tx];
			}

			__syncthreads();

			if (tx == 0)
			{
				hipFree(utility);
				for (int i = 0; i < num_boards; ++i)
				{
					t_v = max(t_utility[tx], t_v);
					if (t_v > beta)
						break;
					alpha = max(alpha, t_v);
				}

				*v = t_v;
			}
			__syncthreads();
		}

		__global__ void master_white_min_kernel(Minimax::utility_type *v, GPUBitBoard const *src, int num_boards, int alpha, int beta, int depth, int turns)
		{
			int tx = threadIdx.x;
			Minimax::utility_type t_v = *v;
			__shared__ Minimax::utility_type *utility;
			__shared__ Minimax::utility_type t_utility[32];

			if (tx == 0)
			{
				hipMalloc(&utility, sizeof(Minimax::utility_type) * num_boards);
				memset(utility, t_v, sizeof(Minimax::utility_type) * num_boards);
			}

			__syncthreads();

			if (tx < num_boards)
			{
				white_max_kernel << <dim3(1, 1, 1), dim3(32, 1, 1) >> > (utility + tx, src[tx], alpha, beta, depth - 1, turns - 1);
				t_utility[tx] = utility[tx];
			}

			__syncthreads();

			if (tx == 0)
			{
				hipFree(utility);
				for (int i = 0; i < num_boards; ++i)
				{
					t_v = min(t_utility[tx], t_v);
					if (t_v < alpha)
						break;
					beta = min(beta, t_v);
				}
				*v = t_v;
			}
			__syncthreads();
		}
	}
}