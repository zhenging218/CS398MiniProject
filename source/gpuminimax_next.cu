#include "hip/hip_runtime.h"
#include "precomp.cuh"
#include "bitboard.h"
#include "gpuminimax.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

namespace Checkers
{
	namespace GPUMinimax
	{
		__global__ void master_white_next_kernel(int *placement, int X, GPUBitBoard const *boards, int num_boards, int depth, int turns)
		{
			int tx = threadIdx.x;
			__shared__ utility_type v[32];
			__shared__ utility_type *ret_v;
			__shared__ hipStream_t streams[4];
			hipEvent_t stream_start, stream_end;
			int t_placement;

			if (tx == 0)
			{
				t_placement = *placement;
				hipMalloc(&ret_v, sizeof(utility_type) * num_boards);
				for (int i = 0; i < 4; ++i)
				{
					hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
				}
			}

			__syncthreads();

			if (tx < num_boards)
			{
				hipEventCreate(&stream_start);
				hipEventCreate(&stream_end);

				int e = tx % 4;
				hipEventRecord(stream_start, streams[e]);
				master_white_min_kernel <<<dim3(1, 1, 1), dim3(32, 1, 1)>>> (ret_v + tx, boards, -Infinity, Infinity, depth, turns);
				hipEventRecord(stream_end, streams[e]);

				hipEventSynchronize(stream_end);
				hipEventDestroy(stream_start);
				hipEventDestroy(stream_end);

				v[tx] = ret_v[tx];
			}

			__syncthreads();

			if (tx == 0)
			{
				// all streams in the block should have completed processing by now.
				for (int i = 0; i < 4; ++i)
				{
					// hipStreamSynchronize(streams[i]);
					hipStreamDestroy(streams[i]);
				}

				for (int i = 0; i < num_boards; ++i)
				{
					if (X < v[i])
					{
						X = v[i];
						t_placement = i;
					}
				}

				*placement = t_placement;
			}

			__syncthreads();
		}

		__global__ void master_black_next_kernel(int *placement, int X, GPUBitBoard const *boards, int num_boards, int depth, int turns)
		{
			int tx = threadIdx.x;
			__shared__ utility_type v[32];
			__shared__ utility_type *ret_v;
			__shared__ hipStream_t streams[4];
			hipEvent_t stream_start, stream_end;
			int t_placement;

			if (tx == 0)
			{
				t_placement = *placement;
				hipMalloc(&ret_v, sizeof(utility_type) * num_boards);
				for (int i = 0; i < 4; ++i)
				{
					hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
				}
			}

			__syncthreads();

			if (tx < num_boards)
			{
				hipEventCreate(&stream_start);
				hipEventCreate(&stream_end);

				int e = tx % 4;
				hipEventRecord(stream_start, streams[e]);
				master_black_min_kernel << <dim3(1, 1, 1), dim3(32, 1, 1) >> > (ret_v + tx, boards, -Infinity, Infinity, depth, turns);
				hipEventRecord(stream_end, streams[e]);

				hipEventSynchronize(stream_end);
				hipEventDestroy(stream_start);
				hipEventDestroy(stream_end);

				v[tx] = ret_v[tx];
			}

			__syncthreads();

			if (tx == 0)
			{
				// all streams in the block should have completed processing by now.
				for (int i = 0; i < 4; ++i)
				{
					// hipStreamSynchronize(streams[i]);
					hipStreamDestroy(streams[i]);
				}

				for (int i = 0; i < num_boards; ++i)
				{
					if (X < v[i])
					{
						X = v[i];
						t_placement = i;
					}
				}

				*placement = t_placement;
			}

			__syncthreads();
		}
	}
}