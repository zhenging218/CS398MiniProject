#include "hip/hip_runtime.h"
#include "precomp.cuh"
#include "bitboard.h"
#include "gpuminimax.h"

namespace Checkers
{
	namespace GPUMinimax
	{
		__global__ void white_next_kernel(int *placement, utility_type *v, utility_type X, GPUBitBoard const *boards, int num_boards, int depth, int turns)
		{
			int tx = threadIdx.x;
			int bx = blockIdx.x;

			__shared__ int frontier_size;
			__shared__ GPUBitBoard frontier[32];
			__shared__ utility_type t_v[32];
			__shared__ utility_type alpha;
			__shared__ utility_type beta;
			__shared__ int gen_board_type;
			__shared__ bool terminated;

			if (tx == 0)
			{
				frontier_size = 0;
				alpha = -Infinity;
				beta = Infinity;
				utility_type terminal_value = 0;
				if (terminated = GetWhiteUtility(boards[bx], terminal_value, depth, turns))
				{
					v[bx] = terminal_value;
				}
				else
				{
					gen_board_type = (GPUBitBoard::GetBlackJumps(boards[bx]) != 0) ? 1 : 0;
				}
			}

			__syncthreads();

			if (!terminated)
			{
				gen_black_move_atomic[gen_board_type](1u << tx, boards[bx], frontier, &frontier_size);

				__syncthreads();

				if (tx < frontier_size)
				{
					t_v[tx] = explore_white_frontier(frontier[tx], alpha, beta, NodeType::MAX, depth - 1, turns - 1);
				}

				__syncthreads();

				// min
				if (tx == 0)
				{
					utility_type t_x = Infinity;
					for(int i = 0; i < frontier_size; ++i)
					{
						t_x = MIN(t_v[i], t_x);
						if (t_x < alpha)
						{
							break;
						}
						beta = MIN(beta, t_x);
					}

					v[bx] = t_x;
				}
			}

			__syncthreads();

			// max
			if (bx == 0 && tx == 0)
			{
				int t_placement = *placement;
				for (int i = 0; i < num_boards; ++i)
				{
					if (X < v[i])
					{
						X = v[i];
						t_placement = i + 1;
					}
				}
				*placement = t_placement;

				
			}

			__syncthreads();
		}

		__global__ void black_next_kernel(int *placement, utility_type *v, utility_type X, GPUBitBoard const *boards, int num_boards, int depth, int turns)
		{
			int tx = threadIdx.x;
			int bx = blockIdx.x;

			__shared__ int frontier_size;
			__shared__ GPUBitBoard frontier[32];
			__shared__ utility_type t_v[32];
			__shared__ utility_type alpha;
			__shared__ utility_type beta;
			__shared__ int gen_board_type;
			__shared__ bool terminated;

			if (tx == 0)
			{
				frontier_size = 0;
				alpha = -Infinity;
				beta = Infinity;
				utility_type terminal_value = 0;
				if (terminated = GetBlackUtility(boards[bx], terminal_value, depth, turns))
				{
					v[bx] = terminal_value;
				}
				else
				{
					gen_board_type = (GPUBitBoard::GetWhiteJumps(boards[bx]) != 0) ? 1 : 0;
				}

			}

			__syncthreads();

			if (!terminated)
			{
				gen_white_move_atomic[gen_board_type](1u << tx, boards[bx], frontier, &frontier_size);   

				__syncthreads();

				if (tx < frontier_size)
				{
					t_v[tx] = explore_black_frontier(frontier[tx], alpha, beta, NodeType::MAX, depth - 1, turns - 1);
				}

				__syncthreads();


				// min
				if (tx == 0)
				{
					utility_type t_x = Infinity;
					for (int i = 0; i < frontier_size; ++i)
					{
						t_x = MIN(t_v[i], t_x);
						if (t_x < alpha)
						{
							break;
						}
						beta = MIN(beta, t_x);
					}

					v[bx] = t_x;
				}
			}

			__syncthreads();

			// max
			if (bx == 0 && tx == 0)
			{
				int t_placement = *placement;
				for (int i = 0; i < num_boards; ++i)
				{
					if (X < v[i])
					{
						X = v[i];
						t_placement = i + 1;
					}
				}

				*placement = t_placement;
			}

			__syncthreads();
		}
	}
}