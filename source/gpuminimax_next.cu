#include "hip/hip_runtime.h"
#include "precomp.cuh"
#include "bitboard.h"
#include "gpuminimax.h"

namespace Checkers
{
	namespace GPUMinimax
	{
		__global__ void white_next_kernel(int *placement, utility_type *v, utility_type X, GPUBitBoard const *boards, int num_boards, int depth, int turns)
		{
			int tx = threadIdx.x;
			int bx = blockIdx.x;

			__shared__ int frontier_size;
			__shared__ GPUBitBoard frontier[32];
			__shared__ utility_type t_v[32];
			__shared__ utility_type alpha;
			__shared__ utility_type beta;
			__shared__ int gen_board_type;
			__shared__ bool terminated;

			if (tx == 0)
			{
				frontier_size = 0;
				alpha = -Infinity;
				beta = Infinity;
				utility_type terminal_value = 0;
				if (terminated = GetWhiteUtility(boards[bx], terminal_value, depth, turns))
				{
					v[bx] = terminal_value;
				}
				else
				{
					gen_board_type = (GPUBitBoard::GetBlackJumps(boards[bx]) != 0) ? 1 : 0;
				}
			}

			__syncthreads();

			if (!terminated)
			{
				gen_black_move_atomic[gen_board_type](1u << tx, boards[bx], frontier, &frontier_size);

				__syncthreads();

				if (tx < frontier_size)
				{
					t_v[tx] = explore_white_frontier(frontier[tx], alpha, beta, NodeType::MAX, depth - 1, turns - 1);
				}

				__syncthreads();

				// min
				for (int i = 1; i < 32; i *= 2)
				{
					if (tx + i < 32)
					{
						t_v[tx] = GET_MIN(t_v[tx], t_v[tx + i]);
					}
				}
				__syncthreads();

				if (tx == 0)
				{
					v[tx] = t_v[tx];
				}
			}

			__syncthreads();

			// max
			if (bx == 0 && tx == 0)
			{
				int t_placement = *placement;
				for (int i = 0; i < num_boards; ++i)
				{
					if (X < v[i])
					{
						X = v[i];
						t_placement = i + 1;
					}
				}
				*placement = t_placement;

				
			}

			__syncthreads();
		}

		__global__ void black_next_kernel(int *placement, utility_type *v, utility_type X, GPUBitBoard const *boards, int num_boards, int depth, int turns)
		{
			int tx = threadIdx.x;
			int bx = blockIdx.x;

			__shared__ int frontier_size;
			__shared__ GPUBitBoard frontier[32];
			__shared__ utility_type t_v[32];
			__shared__ utility_type alpha;
			__shared__ utility_type beta;
			__shared__ int gen_board_type;
			__shared__ bool terminated;

			if (tx == 0)
			{
				frontier_size = 0;
				alpha = -Infinity;
				beta = Infinity;
				utility_type terminal_value = 0;
				if (terminated = GetBlackUtility(boards[bx], terminal_value, depth, turns))
				{
					v[bx] = terminal_value;
				}
				else
				{
					gen_board_type = (GPUBitBoard::GetWhiteJumps(boards[bx]) != 0) ? 1 : 0;
				}

			}

			__syncthreads();

			if (!terminated)
			{
				gen_white_move_atomic[gen_board_type](1u << tx, boards[bx], frontier, &frontier_size);   

				__syncthreads();

				if (tx < frontier_size)
				{
					t_v[tx] = explore_black_frontier(frontier[tx], alpha, beta, NodeType::MAX, depth - 1, turns - 1);
				}

				__syncthreads();


				// min
				// min
				for (int i = 1; i < 32; i *= 2)
				{
					if (tx + i < 32)
					{
						t_v[tx] = GET_MIN(t_v[tx], t_v[tx + i]);
					}
				}
				__syncthreads();

				if (tx == 0)
				{
					v[tx] = t_v[tx];
				}
			}

			__syncthreads();

			// max
			if (bx == 0 && tx == 0)
			{
				int t_placement = *placement;
				for (int i = 0; i < num_boards; ++i)
				{
					if (X < v[i])
					{
						X = v[i];
						t_placement = i + 1;
					}
				}

				*placement = t_placement;
			}

			__syncthreads();
		}
	}
}