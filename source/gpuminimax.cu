#include "hip/hip_runtime.h"
#include "precomp.cuh"
#include "bitboard.h"
#include "gpuminimax.h"

#include <cstdlib>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

namespace Checkers
{
	namespace GPUMinimax
	{
		Minimax::Result Next(BitBoard &board, Minimax::Turn turn, int &depth, int &turns_left)
		{
			if (turns_left == 0)
			{
				return Minimax::DRAW;
			}

			BitBoard frontier[32];

			if (turn == Minimax::WHITE)
			{
				BitBoard *end = frontier;
				BitBoard::GetPossibleWhiteMoves(board, end);
				int size = end - frontier;
				if (size == 0)
				{
					return Minimax::Result::LOSE;
				}

				int placement = -1;
				utility_type X = -Minimax::Infinity;
				utility_type terminal_value = 0;

				// CPU left-most branch
				utility_type v = WhiteMoveMin(frontier[0], depth, turns_left, -Infinity, Infinity);
				if (X < v)
				{
					X = v;
					placement = 0;
				}

				if (size > 1)
				{
					// GPU tree-split the rest of the branches
					GPUBitBoard * GPUFrontier;
					int * GPUPlacement;

					GPUBitBoard *copy = (GPUBitBoard*)malloc(sizeof(GPUBitBoard) * (size - 1));
					for (int i = 0; i < (size - 1); ++i)
					{
						new (copy + i) GPUBitBoard(frontier[i + 1]);
					}
					hipMalloc((void**)&GPUFrontier, sizeof(GPUBitBoard) * (size - 1));
					hipMemcpy(GPUFrontier, copy, sizeof(GPUBitBoard) * (size - 1), hipMemcpyHostToDevice);
					free(copy);

					hipMalloc((void**)&GPUPlacement, sizeof(int));
					hipMemcpy(GPUPlacement, &placement, sizeof(int), hipMemcpyHostToDevice);

					// launch kernel
					master_white_next_kernel << <dim3(((size - 1) / 32) + 1, 1, 1), dim3(32, 1, 1) >> > (GPUPlacement, X, GPUFrontier, size - 1, depth, turns_left);
					hipDeviceSynchronize();

					hipMemcpy(&placement, GPUPlacement, sizeof(int), hipMemcpyDeviceToHost);
					hipFree(GPUFrontier);
					hipFree(GPUPlacement);
				}

				if (placement >= 0)
				{
					board = frontier[placement];
				}
			}
			else
			{
				BitBoard *end = frontier;
				BitBoard::GetPossibleBlackMoves(board, end);
				int size = end - frontier;
				if (size == 0)
				{
					return Minimax::Result::LOSE;
				}

				int placement = -1;
				utility_type X = -Minimax::Infinity;
				utility_type terminal_value = 0;

				// CPU left-most branch
				utility_type v = BlackMoveMin(frontier[0], depth, turns_left, -Infinity, Infinity);
				if (X < v)
				{
					X = v;
					placement = 0;
				}

				if (size > 1)
				{
					// GPU tree-split the rest of the branches
					GPUBitBoard * GPUFrontier;
					int * GPUPlacement;

					GPUBitBoard *copy = (GPUBitBoard*)malloc(sizeof(GPUBitBoard) * (size - 1));
					for (int i = 0; i < (size - 1); ++i)
					{
						new (copy + i) GPUBitBoard(frontier[i + 1]);
					}
					hipMalloc((void**)&GPUFrontier, sizeof(GPUBitBoard) * (size - 1));
					hipMemcpy(GPUFrontier, copy, sizeof(GPUBitBoard) * (size - 1), hipMemcpyHostToDevice);
					free(copy);

					hipMalloc((void**)&GPUPlacement, sizeof(int));
					hipMemcpy(GPUPlacement, &placement, sizeof(int), hipMemcpyHostToDevice);

					// launch kernel
					master_black_next_kernel << <dim3(1, 1, 1), dim3(32, 1, 1) >> > (GPUPlacement, X, GPUFrontier, depth, turns_left);
					hipDeviceSynchronize();

					hipMemcpy(&placement, GPUPlacement, sizeof(int), hipMemcpyDeviceToHost);
					hipFree(GPUFrontier);
					hipFree(GPUPlacement);
				}

				if (placement >= 0)
				{
					board = frontier[placement];
				}
			}

			++turn;
			if (turns_left)
			{
				--turns_left;
			}

			return Minimax::INPROGRESS;
		}

		__host__ utility_type WhiteMoveMax(BitBoard const &b, int depth, int turns_left, utility_type alpha, utility_type beta)
		{
			utility_type v = -Infinity;
			utility_type terminal_value = 0;
			BitBoard frontier[32];
			if (GetWhiteUtility(b, terminal_value, depth, turns_left))
			{
				return terminal_value;
			}

			BitBoard *end = frontier;
			BitBoard::GetPossibleWhiteMoves(b, end);
			int size = end - frontier;

			if (size > 0)
			{

				v = std::max(WhiteMoveMin(frontier[0], depth - 1, turns_left - 1, alpha, beta), v);
				if (!(v > beta)) // if not pruning, then run kernel
				{
					alpha = std::max(alpha, v);
					if (size > 1)
					{
						// GPU tree-split the rest of the branches
						GPUBitBoard * GPUFrontier;
						utility_type * GPUv;

						GPUBitBoard *copy = (GPUBitBoard*)malloc(sizeof(GPUBitBoard) * (size - 1));
						for (int i = 0; i < (size - 1); ++i)
						{
							new (copy + i) GPUBitBoard(frontier[i + 1]);
						}
						hipMalloc((void**)&GPUFrontier, sizeof(GPUBitBoard) * (size - 1));
						hipMemcpy(GPUFrontier, copy, sizeof(GPUBitBoard) * (size - 1), hipMemcpyHostToDevice);
						free(copy);

						hipMalloc((void**)&GPUv, sizeof(utility_type));
						hipMemcpy(GPUv, &v, sizeof(utility_type), hipMemcpyHostToDevice);

						// call master_white_max_kernel because we are in the max function.
						master_white_max_kernel << <dim3(1, 1, 1), dim3(32, 1, 1) >> > (GPUv, GPUFrontier, size - 1, alpha, beta, depth - 1, turns_left - 1);
						hipDeviceSynchronize();
						hipMemcpy(&v, GPUv, sizeof(utility_type), hipMemcpyDeviceToHost);
						hipFree(GPUFrontier);
						hipFree(GPUv);
					}
				}
				return v;
			}
		}

		__host__ utility_type WhiteMoveMin(BitBoard const &b, int depth, int turns_left, utility_type alpha, utility_type beta)
		{
			utility_type v = -Infinity;
			utility_type terminal_value = 0;
			BitBoard frontier[32];
			if (GetWhiteUtility(b, terminal_value, depth, turns_left))
			{
				return terminal_value;
			}

			BitBoard *end = frontier;
			BitBoard::GetPossibleBlackMoves(b, end);
			int size = end - frontier;

			if (size > 0)
			{

				v = std::max(WhiteMoveMax(frontier[0], depth - 1, turns_left - 1, alpha, beta), v);
				if (!(v < alpha)) // if not pruning, then run kernel
				{
					beta = std::min(beta, v);
					if (size > 1)
					{
						// GPU tree-split the rest of the branches
						GPUBitBoard * GPUFrontier;
						utility_type * GPUv;

						GPUBitBoard *copy = (GPUBitBoard*)malloc(sizeof(GPUBitBoard) * (size - 1));
						for (int i = 0; i < (size - 1); ++i)
						{
							new (copy + i) GPUBitBoard(frontier[i + 1]);
						}
						hipMalloc((void**)&GPUFrontier, sizeof(GPUBitBoard) * (size - 1));
						hipMemcpy(GPUFrontier, copy, sizeof(GPUBitBoard) * (size - 1), hipMemcpyHostToDevice);
						free(copy);

						hipMalloc((void**)&GPUv, sizeof(utility_type));
						hipMemcpy(GPUv, &v, sizeof(utility_type), hipMemcpyHostToDevice);

						// call master_white_min_kernel because we are in the min function.
						master_white_min_kernel << <dim3(1, 1, 1), dim3(32, 1, 1) >> > (GPUv, GPUFrontier, size - 1, alpha, beta, depth - 1, turns_left - 1);
						hipDeviceSynchronize();
						hipMemcpy(&v, GPUv, sizeof(utility_type), hipMemcpyDeviceToHost);
						hipFree(GPUFrontier);
						hipFree(GPUv);
					}
				}
				return v;
			}
		}

		__host__ utility_type BlackMoveMax(BitBoard const &b, int depth, int turns_left, utility_type alpha, utility_type beta)
		{

		}

		__host__ utility_type BlackMoveMin(BitBoard const &b, int depth, int turns_left, utility_type alpha, utility_type beta)
		{

		}
	}
}