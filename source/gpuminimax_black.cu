#include "hip/hip_runtime.h"
#include "precomp.cuh"
#include "bitboard.h"
#include "gpuminimax.h"

namespace Checkers
{
	namespace GPUMinimax
	{
		__device__ utility_type black_min_device(GPUBitBoard src, int alpha, int beta, int depth, int turns)
		{
			utility_type v = Infinity;
			utility_type terminal_value = 0;
			int i = 0;
			GPUBitBoard moves[128];
			// check if need to stop the search
			//if (GetBlackUtility(src, terminal_value, depth, turns))
			//	return terminal_value;
			//// 32 possible moves
			//while()
			//	GenBlackMove(i,moves,src)
			//}

			//for (auto const &move : frontier)
			//{
			//	v = min(black_max_device(move, depth - 1, turns - 1, alpha, beta), v);
			//	if (v < alpha)
			//	{
			//		// prune
			//		break;
			//	}
			//	beta = min(beta, v);
			//}

			return v;
		}

		__device__ utility_type black_max_device(GPUBitBoard src, int alpha, int beta, int depth, int turns)
		{
			utility_type v = -Infinity;
			utility_type terminal_value = 0;
			GPUBitBoard moves[128];
			// check if need to stop the search
			if (GetBlackUtility(src, terminal_value, depth, turns))
				return terminal_value;

			// 32 possible moves
			for (int i = 0; i < 32; i++)
			{
				GenBlackMove(i, moves, src)
			}

			for (auto const &move : frontier)
			{
				v = max(BlackMoveMin(move, depth - 1, turns - 1, alpha, beta), v);
				if (v > beta)
				{
					// prune
					break;
				}
				alpha = max(alpha, v);
			}

			return v;
		}
	}
}